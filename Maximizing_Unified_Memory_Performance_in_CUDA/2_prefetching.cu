#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    for (size_t i = 0; i < 6710864 * 8; ++i) {
        managedPtr[i] = static_cast<float>(i);
    }

    int device = 0;
    hipGetDevice(&device);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure prefetching time
    hipEventRecord(start);
    hipMemPrefetchAsync(managedPtr, dataSize, device);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float prefetchTime = 0.0f;
    hipEventElapsedTime(&prefetchTime, start, stop);

    hipEventRecord(start);
    stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float kernelTime = 0.0f;
    hipEventElapsedTime(&kernelTime, start, stop);

    // Calculate bandwidth
    float bandwidth = (dataSize / (prefetchTime+kernelTime)) / 1e6; // GB/s

    std::cout << "Prefetching - Prefetch Bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    return 0;
}
