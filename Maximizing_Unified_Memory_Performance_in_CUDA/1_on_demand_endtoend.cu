#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int numIterations = 100; // Number of iterations
    float totalTime = 0.0f;

    hipEventRecord(start);
    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            managedPtr[j] = static_cast<float>(j+i);
        }
        // Measure kernel execution time for each iteration
        stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&totalTime, start, stop);

    // Calculate average kernel execution time and bandwidth
    float avgTime = totalTime / numIterations;
    std::cout << "Average Time: " << avgTime << " ms" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
