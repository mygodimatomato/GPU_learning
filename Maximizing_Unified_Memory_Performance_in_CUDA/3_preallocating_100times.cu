#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    const int numIterations = 100; // Number of repetitions
    float *hostPtr, *devicePtr, *output;
    hipHostMalloc(&hostPtr, dataSize, hipHostMallocDefault);  // Allocate pinned host memory
    hipMalloc(&devicePtr, dataSize);   // Allocate GPU memory
    hipMalloc(&output, dataSize);

    // for (size_t i = 0; i < 6710864 * 8; ++i) {
    //     hostPtr[i] = static_cast<float>(i);
    // }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTransferTime = 0.0f;
    float totalKernelTime = 0.0f;

    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            hostPtr[j] = static_cast<float>(j+i);
        }
        // Measure data transfer time
        hipEventRecord(start);
        hipMemcpyAsync(devicePtr, hostPtr, dataSize, hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipDeviceSynchronize();

        float transferTime = 0.0f;
        hipEventElapsedTime(&transferTime, start, stop);
        totalTransferTime += transferTime;

        // Measure kernel execution time
        hipEventRecord(start);
        stream_thread<float, READ><<<1, 256>>>(devicePtr, dataSize, output, 0.0f);
        hipEventRecord(stop);
        hipDeviceSynchronize();

        float kernelTime = 0.0f;
        hipEventElapsedTime(&kernelTime, start, stop);
        totalKernelTime += kernelTime;
    }

    float avgTransferTime = totalTransferTime / numIterations;
    float avgKernelTime = totalKernelTime / numIterations;

    // Calculate average bandwidth
    float avgBandwidth = (dataSize / (avgTransferTime + avgKernelTime)) / 1e6; // GB/s

    std::cout << "Average Transfer Time: " << avgTransferTime << " ms" << std::endl;
    std::cout << "Average Kernel Time: " << avgKernelTime << " ms" << std::endl;
    std::cout << "Average Bandwidth: " << avgBandwidth << " GB/s" << std::endl;

    hipHostFree(hostPtr);
    hipFree(devicePtr);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
