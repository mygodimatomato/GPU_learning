#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    // for (size_t i = 0; i < 6710864 * 8; ++i) {
    //     managedPtr[i] = static_cast<float>(i);
    // }

    int device = 0;
    hipGetDevice(&device);

    // Create separate events for prefetch and kernel timing
    hipEvent_t prefetchStart, prefetchStop;
    hipEvent_t kernelStart, kernelStop;
    hipEventCreate(&prefetchStart);
    hipEventCreate(&prefetchStop);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    float totalPrefetchTime = 0.0f;
    float totalKernelTime = 0.0f;
    const int numIterations = 100;

    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            managedPtr[j] = static_cast<float>(j+i);
        }
        // Measure prefetching time
        hipEventRecord(prefetchStart);
        hipMemPrefetchAsync(managedPtr, dataSize, device);
        hipEventRecord(prefetchStop);
        hipDeviceSynchronize();

        float prefetchTime = 0.0f;
        hipEventElapsedTime(&prefetchTime, prefetchStart, prefetchStop);
        totalPrefetchTime += prefetchTime;

        // Measure kernel execution time
        hipEventRecord(kernelStart);
        stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
        hipEventRecord(kernelStop);
        hipDeviceSynchronize();

        float kernelTime = 0.0f;
        hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
        totalKernelTime += kernelTime;
    }

    // Calculate averages
    float avgPrefetchTime = totalPrefetchTime / numIterations;
    float avgKernelTime = totalKernelTime / numIterations;

    // Calculate average bandwidth
    float avgBandwidth = (dataSize / (avgPrefetchTime + avgKernelTime)) / 1e6; // GB/s

    std::cout << "Average Prefetch Time: " << avgPrefetchTime << " ms" << std::endl;
    std::cout << "Average Kernel Execution Time: " << avgKernelTime << " ms" << std::endl;
    std::cout << "Average Bandwidth: " << avgBandwidth << " GB/s" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    hipEventDestroy(prefetchStart);
    hipEventDestroy(prefetchStop);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelStop);

    return 0;
}
