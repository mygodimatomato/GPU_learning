#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    for (size_t i = 0; i < 6710864 * 8; ++i) {
        managedPtr[i] = static_cast<float>(i);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure kernel execution time
    hipEventRecord(start);
    stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float kernelTime = 0.0f;
    hipEventElapsedTime(&kernelTime, start, stop);

    // Calculate bandwidth
    float bandwidth = (dataSize / kernelTime) / 1e6; // GB/s

    std::cout << "On-demand Migration - Kernel Bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    return 0;
}
