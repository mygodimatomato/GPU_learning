#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    int device = 0;
    hipGetDevice(&device);

    // Create separate events for prefetch and kernel timing
    hipEvent_t Start, Stop;
    hipEventCreate(&Start);
    hipEventCreate(&Stop);

    float totalTime = 0.0f;
    const int numIterations = 100;

    hipEventRecord(Start);
    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            managedPtr[j] = static_cast<float>(j+i);
        }
        hipMemPrefetchAsync(managedPtr, dataSize, device);

        stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
    }
    hipEventRecord(Stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&totalTime, Start, Stop);
    // Calculate averages
    float avgTime = totalTime / numIterations;

    std::cout << "Average Time: " << avgTime << " ms" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    hipEventDestroy(Start);
    hipEventDestroy(Stop);

    return 0;
}
