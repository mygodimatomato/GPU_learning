#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}


int main() {
    const size_t dataSize = 1024 * sizeof(float);
    float *hostPtr, *devicePtr, *output;
    hipHostMalloc(&hostPtr, dataSize, hipHostMallocDefault);  // Allocate pinned host memory
    hipMalloc(&devicePtr, dataSize);   // Allocate GPU memory
    hipMalloc(&output, dataSize);

    // Initialize host memory
    for (size_t i = 0; i < 1024; ++i) {
        hostPtr[i] = static_cast<float>(i);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure data transfer time
    hipEventRecord(start);
    hipMemcpyAsync(devicePtr, hostPtr, dataSize, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float transferTime = 0.0f;
    hipEventElapsedTime(&transferTime, start, stop);

    // Measure kernel execution time
    hipEventRecord(start);
    stream_thread<float, READ><<<1, 256>>>(devicePtr, dataSize, output, 0.0f);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float kernelTime = 0.0f;
    hipEventElapsedTime(&kernelTime, start, stop);

    std::cout << "Explicit Data Transfer - Transfer Time: " << transferTime << " ms, Kernel Time: " << kernelTime << " ms" << std::endl;
    std::cout << "Total bandwidth: " << (dataSize / (transferTime + kernelTime)) / 1e6 << " GB/s" << std::endl; 

    hipHostFree(hostPtr);
    hipFree(devicePtr);
    hipFree(output);
    return 0;
}
