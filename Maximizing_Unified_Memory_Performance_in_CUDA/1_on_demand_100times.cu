#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    // for (size_t i = 0; i < 6710864 * 8; ++i) {
    //     managedPtr[i] = static_cast<float>(i);
    // }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int numIterations = 100; // Number of iterations
    float totalKernelTime = 0.0f;

    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            managedPtr[j] = static_cast<float>(j+i);
        }
        // Measure kernel execution time for each iteration
        hipEventRecord(start);
        stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
        hipEventRecord(stop);
        hipDeviceSynchronize();

        float kernelTime = 0.0f;
        hipEventElapsedTime(&kernelTime, start, stop);

        // Accumulate total kernel time
        totalKernelTime += kernelTime;
    }

    // Calculate average kernel execution time and bandwidth
    float avgKernelTime = totalKernelTime / numIterations;
    float avgBandwidth = (dataSize / avgKernelTime) / 1e6; // GB/s

    std::cout << "Average Kernel Execution Time: " << avgKernelTime << " ms" << std::endl;
    std::cout << "Average Bandwidth: " << avgBandwidth << " GB/s" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
