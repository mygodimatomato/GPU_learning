#include <hip/hip_runtime.h>
#include <iostream>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 6710864 * 8 * sizeof(float); // Size in bytes
    const int numIterations = 100; // Number of repetitions
    float *hostPtr, *devicePtr, *output;
    hipHostMalloc(&hostPtr, dataSize, hipHostMallocDefault);  // Allocate pinned host memory
    hipMalloc(&devicePtr, dataSize);   // Allocate GPU memory
    hipMalloc(&output, dataSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < numIterations; ++i) {
        for (size_t j = 0; j < 6710864 * 8; ++j) {
            hostPtr[j] = static_cast<float>(j+i);
        }
        hipMemcpyAsync(devicePtr, hostPtr, dataSize, hipMemcpyHostToDevice);
        stream_thread<float, READ><<<1, 256>>>(devicePtr, dataSize, output, 0.0f);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    float totalTime = 0.0f;
    hipEventElapsedTime(&totalTime, start, stop);

    float averageTime = totalTime / numIterations;

    std::cout << "Average Time: " << averageTime << " ms" << std::endl;


    hipHostFree(hostPtr);
    hipFree(devicePtr);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
