#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define READ 0
#define WRITE 1

template <typename data_type, int op>
__global__ void stream_thread(data_type *ptr, const size_t size, 
                              data_type *output, const data_type val) 
{ 
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x; 
  size_t n = size / sizeof(data_type); 
  data_type accum = 0; 

  for(; tid < n; tid += blockDim.x * gridDim.x) 
    if (op == READ) accum += ptr[tid]; 
      else ptr[tid] = val;  

  if (op == READ) 
    output[threadIdx.x + blockIdx.x * blockDim.x] = accum; 
}

int main() {
    const size_t dataSize = 1024 * sizeof(float);
    float *managedPtr, *output;
    hipMallocManaged(&managedPtr, dataSize);
    hipMallocManaged(&output, dataSize);

    // Initialize managed memory
    for (size_t i = 0; i < 1024; ++i) {
        managedPtr[i] = static_cast<float>(i);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure kernel execution time
    hipEventRecord(start);
    stream_thread<float, READ><<<1, 256>>>(managedPtr, dataSize, output, 0.0f);
    hipEventRecord(stop);
    hipDeviceSynchronize();

    float kernelTime = 0.0f;
    hipEventElapsedTime(&kernelTime, start, stop);

    std::cout << "On-demand Migration - Kernel Time: " << kernelTime << " ms" << std::endl;
    std::cout << "Total bandwidth: " << (dataSize / kernelTime) / 1e6 << " GB/s" << std::endl;

    hipFree(managedPtr);
    hipFree(output);
    return 0;
}
