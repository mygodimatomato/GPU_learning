#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total Global Memory (GB): %f\n",
           prop.totalGlobalMem/1.0e9);
    printf("  Shared Memory per Block (KB): %d\n",prop.sharedMemPerBlock/1024);
    printf("  Registers per Block: %d\n", prop.regsPerBlock);
    printf("  Warp Size: %d\n", prop.warpSize);
    printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("  Max Threads per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  Max Threads per Block Dimension: %d x %d x %d\n",
           prop.maxThreadsDim[0],
           prop.maxThreadsDim[1],
           prop.maxThreadsDim[2]);

  }
}