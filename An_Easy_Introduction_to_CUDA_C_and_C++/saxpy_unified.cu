
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  } 
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  printf("Max error: %f\n", maxError);

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}