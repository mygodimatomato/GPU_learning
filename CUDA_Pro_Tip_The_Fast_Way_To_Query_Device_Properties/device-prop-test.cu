// Compiling and running this program:
//   nvcc -std=c++11 device-prop-test.cu && ./a.out

#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
using namespace std;

#define CUDA_CHECK(call)                                    \
  do {                                                      \
    hipError_t status = call;                              \
    if(status != hipSuccess) {                             \
      printf("FAIL: call='%s'. Reason:%s\n", #call,         \
             hipGetErrorString(status));                   \
      return -1;                                            \
    }                                                       \
  } while (0)

int main(int argc, char** argv) {
  int devId;
  CUDA_CHECK(hipGetDevice(&devId));

  auto start = chrono::high_resolution_clock::now();
  hipDeviceProp_t prop;
  for(int i = 0; i < 25; ++i) {
    CUDA_CHECK(hipGetDeviceProperties(&prop, devId));
  }
  auto end = chrono::high_resolution_clock::now();
  cout
    << "hipGetDeviceProperties -> "
    << chrono::duration_cast<chrono::microseconds>(end - start).count() / 25.0
    << "us" << endl;

  int smemSize, numProcs;
  start = chrono::high_resolution_clock::now();
  for(int i = 0; i < 25; ++i) {
    CUDA_CHECK(hipDeviceGetAttribute(&smemSize,
                                      hipDeviceAttributeMaxSharedMemoryPerBlock,
                                      devId));
    CUDA_CHECK(hipDeviceGetAttribute(&numProcs,
                                      hipDeviceAttributeMultiprocessorCount,
                                      devId));
  }
  end = chrono::high_resolution_clock::now();
  cout
    << "hipDeviceGetAttribute -> "
    << chrono::duration_cast<chrono::microseconds>(end - start).count() / 25.0
    << "us" << endl;
  return 0;
}