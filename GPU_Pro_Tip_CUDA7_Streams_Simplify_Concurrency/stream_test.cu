#include <stdio.h>              // for printf (if you want to print)
#include <math.h>               // for sqrt, pow
#include <hip/hip_runtime.h>       // main CUDA runtime API
  // needed on some compilers for block/thread indexing

// Example array size: 1M elements
const int N = 1 << 20;  // 1,048,576

// Simple kernel that writes sqrt(pi^i) for each index
__global__ void kernel(float *x, int n)
{
    // Compute global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Stride loop to cover all elements
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // For numeric stability and performance, cast constants to float
        x[i] = sqrtf(powf(3.14159f, (float)i));
    }
}
int main()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    return 0;
}