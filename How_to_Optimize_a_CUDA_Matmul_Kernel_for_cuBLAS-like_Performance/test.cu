// matmul_kernel.cu
// Compute C = alpha * A * B + beta * C
#include <hip/hip_runtime.h>

// tweak this tile size as you optimize
#ifndef TILE_SIZE
#define TILE_SIZE 16
#endif

extern "C"
__global__ void matmul_kernel(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float* C,
    int N,
    float alpha,
    float beta
) {
    // global row/col index for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // guard against threads outside the N×N matrix
    if (row >= N || col >= N) return;

    // accumulator for the dot product
    float sum = 0.0f;

    // === Naïve version ===
    for (int k = 0; k < N; ++k) {
        sum += A[row * N + k] * B[k * N + col];
    }

    // writeback with alpha/beta scaling
    C[row * N + col] = alpha * sum + beta * C[row * N + col];

    /* === Tiled version stub (optional next step) ===
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    sum = 0.0f;
    int numTiles = (N + TILE_SIZE - 1) / TILE_SIZE;
    for (int t = 0; t < numTiles; ++t) {
        int tiledCol = t * TILE_SIZE + threadIdx.x;
        int tiledRow = t * TILE_SIZE + threadIdx.y;
        As[threadIdx.y][threadIdx.x] =
            (row < N && tiledCol < N) ? A[row * N + tiledCol] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] =
            (tiledRow < N && col < N) ? B[tiledRow * N + col] : 0.0f;
        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }
        __syncthreads();
    }
    C[row * N + col] = alpha * sum + beta * C[row * N + col];
    */
}
