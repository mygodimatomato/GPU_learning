#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define MAX_BLOCKS 65535

__global__ void device_copy_scalar_kernel(int* d_in, int* d_out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        d_out[i] = d_in[i];
    }
}

void device_copy_scalar(int* d_in, int* d_out, int N) {
    int threads = 128;
    int blocks = min((N + threads - 1) / threads, MAX_BLOCKS);
    device_copy_scalar_kernel<<<blocks, threads>>>(d_in, d_out, N);
}

int main() {
    const int N = 4096;
    int *h_in = new int[N];
    int *h_out = new int[N];

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_in[i] = i;
    }

    // Device memory pointers
    int *d_in, *d_out;

    // Allocate device memory
    hipMalloc((void**)&d_in, N * sizeof(int));
    hipMalloc((void**)&d_out, N * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    // Call the device copy function
    device_copy_scalar(d_in, d_out, N);

    // Copy data back from device to host
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify the output
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (h_out[i] != h_in[i]) {
            success = false;
            break;
        }
    }

    if (success) {
        std::cout << "Data copied successfully!" << std::endl;
    } else {
        std::cout << "Data copy failed." << std::endl;
    }

    // Free device and host memory
    hipFree(d_in);
    hipFree(d_out);
    delete[] h_in;
    delete[] h_out;

    return 0;
}
