#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

#define MAX_BLOCKS 65535

__global__ void device_copy_vector4_kernel(int* d_in, int* d_out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < N/4; i += blockDim.x * gridDim.x) {
        reinterpret_cast<int4*>(d_out)[i] = reinterpret_cast<int4*>(d_in)[i];
    }

    // in only one thread, process final element(if there is one)
    int remainder = N % 4;
    if (idx==N/4 && remainder != 0) {
      while(remainder) {
        int idx = N - remainder--;
        d_out[idx] = d_in[idx];
      }
    }
}

void device_copy_vector4(int* d_in, int* d_out, int N) {
    int threads = 128;
    int blocks = min((N + threads - 1) / threads, MAX_BLOCKS);
    // int blocks = (N + threads - 1) / threads;

    // Measure kernel execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    device_copy_vector4_kernel<<<blocks, threads>>>(d_in, d_out, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate and print bandwidth
    float gb = (2.0f * N * sizeof(int)) / (1e9); // 2xN because both input and output are accessed
    float bandwidth = gb / (milliseconds / 1000.0f); // GB/s
    std::cout << "Array size: " << N << ", Kernel execution time: " << milliseconds << " ms, Bandwidth: " << bandwidth << " GB/s" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    std::vector<int> sizes;
    for (int size = 4096; size <= 536870912; size *= 2) {
        sizes.push_back(size);
    }

    for (int N : sizes) {
        int *h_in = new int[N];
        int *h_out = new int[N];

        // Initialize input data
        for (int i = 0; i < N; i++) {
            h_in[i] = i;
        }

        // Device memory pointers
        int *d_in, *d_out;

        // Allocate device memory
        hipMalloc((void**)&d_in, N * sizeof(int));
        hipMalloc((void**)&d_out, N * sizeof(int));

        // Measure data transfer time from host to device
        auto start_h2d = std::chrono::high_resolution_clock::now();
        hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);
        auto end_h2d = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> h2d_time = end_h2d - start_h2d;

        std::cout << "Array size: " << N << ", Host to Device transfer time: " << h2d_time.count() << " seconds" << std::endl;

        // Call the device copy function
        device_copy_vector4(d_in, d_out, N);

        // Measure data transfer time from device to host
        auto start_d2h = std::chrono::high_resolution_clock::now();
        hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
        auto end_d2h = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> d2h_time = end_d2h - start_d2h;

        std::cout << "Array size: " << N << ", Device to Host transfer time: " << d2h_time.count() << " seconds" << std::endl;

        // Verify the output
        bool success = true;
        for (int i = 0; i < N; i++) {
            if (h_out[i] != h_in[i]) {
                success = false;
                break;
            }
        }

        if (success) {
            std::cout << "Array size: " << N << ", Data copied successfully!" << std::endl;
        } else {
            std::cout << "Array size: " << N << ", Data copy failed." << std::endl;
        }

        // Free device and host memory
        hipFree(d_in);
        hipFree(d_out);
        delete[] h_in;
        delete[] h_out;
    }

    return 0;
}
